
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <pwd.h>
#include <float.h>

#define N 40
#define THREADS 10

__global__ void findmax(float *A, float *max)
{
   __shared__ float sdata[THREADS];
   int i = blockDim.x*blockIdx.x + threadIdx.x, half;

   if (i < N) {
      sdata[threadIdx.x] = A[i];
   } else {
      sdata[threadIdx.x] = FLT_MIN;
   }

   half = 1<<(int)(log2((float)(blockDim.x - 1)));
   for (unsigned int s = half; s > 0; s>>=1) {
      if (threadIdx.x + s < blockDim.x) {
	 if (threadIdx.x < s) {
	    if (sdata[threadIdx.x] < sdata[threadIdx.x + s]) {
	       sdata[threadIdx.x] = sdata[threadIdx.x + s];
	    }
	 }
      }
      __syncthreads();
   }

   if (threadIdx.x == 0) max[blockIdx.x] = sdata[0];
}

int main()
{
   float A[N], *A_d, *max_arr, *max_arr_d, max;
   int i, dev;

   dim3 dimBlock(THREADS);
   dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);

   dev = (getpwuid(getuid()) -> pw_name[3] - '0') % 2 ? 1 : 0;
   hipSetDevice(dev);

   srand(1);
   for (i = 0; i < N; i++) {
      A[i] = rand() % 999;
      printf("%2.1f ", A[i]);
   }
   printf("\n");

   hipMalloc((void **) &A_d, sizeof(float) * N);
   hipMemcpy(A_d, A, sizeof(float) * N, hipMemcpyHostToDevice);

   hipMalloc((void **) &max_arr_d, dimGrid.x * sizeof(float));

   findmax<<<dimGrid, dimBlock>>>(A_d, max_arr_d);
   
   max_arr = (float *)malloc(dimGrid.x * sizeof(float));
   hipMemcpy(max_arr, max_arr_d, dimGrid.x * sizeof(float), hipMemcpyDeviceToHost);

   max = max_arr[0];
   for (i = 1; i < dimGrid.x; i++) {
      if (max < max_arr[i]) max = max_arr[i];
   }
   printf("%f\n", max);

   hipFree(A_d);
   hipFree(max_arr_d);
   free(max_arr);
   exit(0);
}
