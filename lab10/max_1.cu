
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <pwd.h>

#define N 10
#define THREADS 10

__global__ void findmax(float *A, float *max)
{
   __shared__ float sdata[THREADS];
   int i = blockDim.x*blockIdx.x + threadIdx.x, half;

   sdata[threadIdx.x] = A[i];

   half = blockDim.x;
   for (unsigned int s = half; s > 0; s >>= 1) {
      
      if (threadIdx.x < s) {
	 if (sdata[threadIdx.x] < sdata[threadIdx.x + s]) {
	    sdata[threadIdx.x] = sdata[threadIdx.x + s];
	 }
      }

      __syncthreads();
   }

   if (threadIdx.x == 0) *max = sdata[0];
}

int main()
{
   float A[N], *A_d, max, *max_d;
   int i, dev;

   dim3 dimBlock(THREADS);
   dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);

   dev = (getpwuid(getuid())->pw_name[3]-'0')%2? 1: 0;
   hipSetDevice(dev);

   srand(1);
   for (i = 0; i < N; i++) {
      A[i] = rand() % 999;
      printf("%2.1f ", A[i]);
   }
   printf("\n");

   hipMalloc((void **) &A_d, sizeof(float) * N);
   hipMemcpy(A_d, A, sizeof(float) * N, hipMemcpyHostToDevice);

   hipMalloc((void **) &max_d, sizeof(float));

   findmax<<<dimGrid, dimBlock>>>(A_d, max_d);
   
   hipMemcpy(&max, max_d, sizeof(float), hipMemcpyDeviceToHost);

   printf("%f\n", max);

   hipFree(A_d);
   hipFree(max_d);
   exit(0);
}
