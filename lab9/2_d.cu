
#include <hip/hip_runtime.h>
#include <stdio.h>

#define X 9
#define Y 8
#define THREAD_X 3
#define THREAD_Y 2

#define A(i,j) A[i*Y+j]

__global__ void index(int *A)
{
   int i = blockDim.x*blockIdx.x+threadIdx.x;
   int j = blockDim.y* blockIdx.y+threadIdx.y;
   
   //(1) 
   A(i,j) = threadIdx.x;
   //(2) 
   A(i,j) = threadIdx.y;
   //(3) 
   A(i,j) = blockIdx.y;
   //(4) 
   A(i,j) = blockIdx.y;
   //(5) 
   A(i,j) = gridDim.x;
   //(6) 
   A(i,j) = gridDim.y;
   //(7) 
   A(i,j) = blockDim.x;
   //(8) 
   A(i,j) = blockDim.y;
   //(9) 
   A(i,j) = i;
   //(10) 
   A(i,j) = j;
   //(11) 
   A(i,j) = i*Y+j;
}

int main()
{
   int A[X][Y], *A_d;
   int i, j;
   
   dim3 dimBlock(THREAD_X,THREAD_Y);
   dim3 dimGrid(X/dimBlock.x, Y/dimBlock.y);
   
   hipSetDevice(0); // or 1
   hipMalloc((void **)&A_d, sizeof(int)*X*Y);
   
   for (i=0; i<X; i++)
      for (j=0; j<Y; j++)
	 A[i][j] = 1;

   hipMemcpy(A_d, A, sizeof(int)*X*Y, hipMemcpyHostToDevice);

   index<<<dimGrid, dimBlock>>>(A_d);

   hipMemcpy(A, A_d, sizeof(int)*X*Y, hipMemcpyDeviceToHost);

   
   for (i=0; i<X; i++) {
      for (j=0; j<Y; j++) {
	 printf("%3d ", A[i][j]);
      }
      printf("\n");
   }
   hipFree(A_d);
}
