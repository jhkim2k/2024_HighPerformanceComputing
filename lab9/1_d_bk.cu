
#include <hip/hip_runtime.h>
#include <stdio.h>
#define X 12
#define THREAD_X 4
__global__ void index(int *A)
{
   int i = blockDim.x*blockIdx.x+threadIdx.x;
   //(1) 
   A[i] = 123;
   //(2) 
   A[i] = i;
   //(3) 
   A[i] = gridDim.x;
   //(4) 
   A[i] = blockDim.x;
   //(5) 
   A[i] = threadIdx.x;
}
int main()
{
   int A[X], *A_d, i;
   dim3 dimBlock(THREAD_X);
   dim3 dimGrid(X/ dimBlock.x);
   
   hipSetDevice(0); // or 1
   hipMalloc((void **)&A_d, sizeof(int)*X);
   for (i=0; i<X; i++)
      A[i] = 1;
   
   hipMemcpy(A_d, A, sizeof(int)*X, hipMemcpyHostToDevice);
   
   index<<<dimGrid, dimBlock>>>(A_d);
	
   hipMemcpy(A, A_d, sizeof(int)*X, hipMemcpyDeviceToHost);

   for (i=0; i<X; i++)
      printf("%d ", A[i]);
   printf("\n");
   hipFree(A_d);
}
