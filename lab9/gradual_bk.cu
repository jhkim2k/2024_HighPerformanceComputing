
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 12

#define A(i,j) A[i*N+j]

__global__ void gradual(int *A)
{
   int j = blockDim.y * blockIdx.y + threadIdx.y;
   int i = blockDim.x * blockIdx.x + threadIdx.x;

   if (j < N && i < N) {
      if (j <= i) {
	 A(i, j) = i;
      } else {
	 A(i, j) = j;
      }
   }
}

int main(int argc, char *argv[])
{
   int A[N][N], *A_d;
   int nt, i, j, dev;

   // GPU info
   hipDeviceProp_t deviceProp;

   dev = 0;
   hipSetDevice(dev);
   hipGetDevice(&dev);
   hipGetDeviceProperties(&deviceProp, dev);
   printf("Using Device %d: \"%s\"\n", dev, deviceProp.name);

   if (argc != 2) {
      fprintf(stderr, "usage: %s #threads\n", argv[0]);
      exit(0);
   }
   nt = atoi(argv[1]);

   dim3 dimBlock(nt, nt);
   dim3 dimGrid((N + dimBlock.x - 1)/dimBlock.x, (N + dimBlock.y - 1)/dimBlock.y);

   hipMalloc((void **) &A_d, sizeof(float)*N*N);

   hipMemcpy(A_d, A, N*N*sizeof(float), hipMemcpyHostToDevice);
   
   gradual<<<dimGrid, dimBlock>>>(A_d);

   hipMemcpy(A, A_d, N*N*sizeof(float), hipMemcpyDeviceToHost);

   for (i = 0; i < N; i++) {
      for (j = 0; j < N; j++) {
	 printf("%3d ", A[i][j]);
      }
      printf("\n");
   }

   hipFree(A_d);
   return 0;
}
