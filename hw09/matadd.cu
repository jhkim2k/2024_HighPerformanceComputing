
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 12
#define N 10
#define A(i, j) A[i * N + j]
#define B(i, j) B[i * N + j]
#define C(i, j) C[i * N + j]

__global__ void matadd(float *A, float *B, float *C)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = blockIdx.y * blockDim.y + threadIdx.y;

   if (i < M && j < N) {
      C(i, j) = A(i, j) + B(i, j);
   }
}

int main(int argc, char *argv[])
{
   float A[M][N], B[M][N], C[M][N];
   float *A_d, *B_d, *C_d;
   int THREADS_M, THREADS_N, i, j;

   if (argc != 3) {
      fprintf(stderr, "usage: %s threads_M threads_N\n", argv[0]);
      exit(0);
   }
   THREADS_M = atol(argv[1]);
   THREADS_N = atoi(argv[2]);

   dim3 dimBlock(THREADS_M, THREADS_N);
   dim3 dimGrid((M + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);

   for (i = 0; i < M; i++) {
      for (j = 0; j < N; j++) {
	 A[i][j] = M - i  + 1;
	 B[i][j] = N + j;
      }
   }

   hipMalloc((void **)&A_d, sizeof(float)*M*N);
   hipMalloc((void **)&B_d, sizeof(float)*M*N);
   hipMalloc((void **)&C_d, sizeof(float)*M*N);

   hipMemcpy(A_d, A, sizeof(float)*M*N, hipMemcpyHostToDevice);
   hipMemcpy(B_d, B, sizeof(float)*M*N, hipMemcpyHostToDevice);

   matadd<<<dimGrid, dimBlock>>>(A_d, B_d, C_d);

   hipMemcpy(C, C_d, sizeof(float)*M*N, hipMemcpyDeviceToHost);

   for (i = 0; i < M; i++) {
      for (j = 0; j < N; j++) {
	 printf("%3.0f ", C[i][j]);
      }
      printf("\n");
   }

   hipFree(A_d);
   hipFree(B_d);
   hipFree(C_d);
   exit(0);
}
