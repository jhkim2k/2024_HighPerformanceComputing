
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <pwd.h>

#define MAX_THREADS 256

#define gflops(n, ms) (((n * 18.0) / (ms / 1.0e+3)) / 1.0e+9)

__device__ double f(double x)
{
   // 사다리꼴 적분을 위한 함수: f(x) = 9 * x / (x^3 + 1)^2
   double temp = x * x * x + 1;
   return (9 * x) / (temp * temp);
}

__global__ void area_kernel(double* local_area, long N, double a, double b)
{
   double dx = (b - a) / (double)N;
   long i = blockDim.x * blockIdx.x + threadIdx.x;
   extern __shared__ double sdata[];

   // 각 스레드는 x 좌표에서의 면적을 계산
   double x = a + i * dx;
   // sdata[threadIdx.x] = (i < N) ? 0.5 * (f(x) + f(x + dx)) * dx : 0.0;
   if (i < N) {
      sdata[threadIdx.x] = 0.5 * (f(x) + f(x + dx)) * dx;
   } else {
      sdata[threadIdx.x] = 0.0;
   }

   __syncthreads();

   // 공유 메모리를 사용하여 블록 내에서 부분 합을 계산
   for (int half = blockDim.x / 2; half > 0; half >>= 1) {
      if (threadIdx.x < half) {
	 sdata[threadIdx.x] += sdata[threadIdx.x + half];
      }
      __syncthreads();
   }

   // 블록의 첫 번째 스레드가 결과를 기록
   if (threadIdx.x == 0) {
      local_area[blockIdx.x] = sdata[0];
   }
}

int main(int argc, char* argv[])
{
   double *local_area, *local_area_d, area = 0.0, a = 0.0, b = 2.0;
   long N;
   int nt, i, smsize, dev;
   hipEvent_t start, stop;
   float elapsed;
   hipDeviceProp_t deviceProp;

   if (argc != 3) {
      fprintf(stderr, "usage: %s #segments #threads \n", argv[0]);
      exit(1);
   }
   N = atol(argv[1]);
   nt = atoi(argv[2]);

   if (nt > MAX_THREADS) {
      fprintf(stderr, "%d threads are used. \n", MAX_THREADS);
      nt = MAX_THREADS;
   }

   dim3 dimBlock(nt);
   dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);
   smsize = sizeof(double) * nt;

   // 현재 사용자와 디바이스 ID를 바탕으로 GPU 선택
   dev = (getpwuid(getuid())->pw_name[3] - '0') % 2 ? 1 : 0;
   hipSetDevice(dev);
   hipGetDeviceProperties(&deviceProp, dev);
   printf("Device(%d) used: \"%s\"\n", dev, deviceProp.name);

   local_area = (double*)malloc(sizeof(double) * dimGrid.x);
   hipMalloc((void**)&local_area_d, sizeof(double) * dimGrid.x);

   // 타이머 시작: 계산 시간 측정
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);

   // GPU 커널 호출
   area_kernel<<<dimGrid, dimBlock, smsize>>>(local_area_d, N, a, b);

   // GPU에서 CPU로 결과 복사
   hipMemcpy(local_area, local_area_d, sizeof(double) * dimGrid.x, hipMemcpyDeviceToHost);

   // 타이머 종료: 시간 측정 완료
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsed, start, stop);

   // GPU에서 계산된 부분 합을 CPU에서 최종 합산
   for (i = 0; i < dimGrid.x; i++) {
      area += local_area[i];
   }

   printf("area: %5.5lf\n", area);
   printf("elapsed time: %5.2f milliseconds", elapsed);
   printf(" (GFLOPS: %5.2f)\n", gflops(N, elapsed));

   free(local_area);
   hipFree(local_area_d);
   exit(0);
}

