
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <pwd.h>

#define MAX_THREADS 256

#define gflops(n,ms) (((n*18.0)/(ms/1.0e+3))/1.0e+9)

__device__ double f(double x)
{
   double temp = x * x * x + 1;
   return ((9 * x) / (temp * temp));
}

__global__ void area_kernel(double *local_area, long N, double a, double b)
{
   double dx, x;
   long i = blockDim.x * blockIdx.x + threadIdx.x;
   int half;
   extern __shared__ double sdata[];
   unsigned int s = 0;

   dx = (b - a) / (double)N;
   x = a + (double)i * dx;

   if (i < N) {
      sdata[threadIdx.x] = 0.5 * (f(x) + f(x + dx)) * dx;
   } else {
      sdata[threadIdx.x] = 0.0;
   }

   half = 1<<(int)(log2((double)(blockDim.x-1)));
   for (s = half; s > 0; s >>= 1) {
      if (threadIdx.x + s < blockDim.x) {
	 if (threadIdx.x < s) {
	    sdata[threadIdx.x] += sdata[threadIdx.x + s];
	 }
      }
      __syncthreads();
   }

   if (threadIdx.x == 0) local_area[blockIdx.x] = sdata[0];
}

int main(int argc, char *argv[])
{
   double *local_area, *local_area_d, area, a, b;
   long N;
   int nt, i, smsize, dev;
   hipEvent_t start, stop;
   float elapsed;
   hipDeviceProp_t deviceProp;

   if (argc != 3) {
      fprintf(stderr, "usage: %s #segments #threads \n", argv[0]);
      exit(1);
   }
   
   area = 0.0;
   a = 0.0;
   b = 2.0;

   N = atol(argv[1]);
   nt = atoi(argv[2]);

   if (nt > MAX_THREADS) {
      fprintf(stderr, "%d threads are used. \n", MAX_THREADS);
      nt = MAX_THREADS;
   }

   dim3 dimBlock(nt);
   dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);
   smsize = sizeof(double) * nt;

   dev = (getpwuid(getuid())->pw_name[3]-'0')%2? 1: 0;
   hipSetDevice(dev);
   hipGetDeviceProperties(&deviceProp, dev);
   printf("Device(%d) used: \"%s\"\n", dev, deviceProp.name);

   local_area = (double *)malloc(sizeof(double) * dimGrid.x);
   hipMalloc((void **)&local_area_d, sizeof(double) * dimGrid.x);

   // timing:start
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);

   // call GPU kernel function
   area_kernel<<<dimGrid, dimBlock, smsize>>>(local_area_d, N, a, b);

   // copy values from GPU memory to CPU memory
   hipMemcpy(local_area, local_area_d, sizeof(double) * dimGrid.x, hipMemcpyDeviceToHost);

   // timing:stop
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsed, start, stop);

   //sum local_area's computed on GPU
   for (i = 0; i < dimGrid.x; i++) {
      area += local_area[i];
   }

   printf("area: %5.5lf\n", area);
   printf("elapsed time: %5.2f miliseconds", elapsed);
   printf(" (GFLOPS: %5.2f)\n", gflops(N, elapsed));
   //printf(" (GFLOPS: %5.2f)\n", (((N*18.0)/(elapsed/1.0e + 3))/1.0e + 9));

   free(local_area);
   hipFree(local_area_d);

   exit(0);
}
